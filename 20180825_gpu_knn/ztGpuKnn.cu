#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "hip/device_functions.h"

#include <stdio.h>
#include <string.h>
#include "ztGpuKnn.h"

/*********************** gpuָ�붨�� ***********************/
int   *_gpuKdTreeIndex;
float *_gpuKdTreeData;
float *_gpuSearchPoint;
int *_gpuSearchResult;

int *_gpuKNeighbors;
float *_gpuKDistances;
/***********************************************************/

int initCudaForKdtree(int n, int dim, int nn, int *index, float *treeData)
{
	if (hipMalloc((void **)&_gpuKdTreeIndex, 4 * n * sizeof(int)) != hipSuccess)
	{
		return 1;
	}

	if (hipMemcpy(_gpuKdTreeIndex, index, 4 * n * sizeof(int), hipMemcpyHostToDevice) != hipSuccess)
	{
		return 1;
	}

	if (hipMalloc((void **)&_gpuKdTreeData, n * dim * sizeof(float)) != hipSuccess)
	{
		return 1;
	}

	if (hipMemcpy(_gpuKdTreeData, treeData, n * dim * sizeof(float), hipMemcpyHostToDevice) != hipSuccess)
	{
		return 1;
	}

	if (hipMalloc((void **)&_gpuSearchPoint, ALLTHREADS * dim * sizeof(float)) != hipSuccess)
	{
		return 1;
	}

	if (hipMalloc((void **)&_gpuSearchResult, ALLTHREADS * nn * sizeof(int)) != hipSuccess)
	{
		return 1;
	}

	if (hipMalloc((void **)&_gpuKNeighbors, ALLTHREADS * nn * sizeof(int)) != hipSuccess)
	{
		return 1;
	}

	if (hipMalloc((void **)&_gpuKDistances, ALLTHREADS * nn * sizeof(float)) != hipSuccess)
	{
		return 1;
	}

	return 0;
}

__global__ void gpuSearchKernal(int *gpuKN, float *gpuKD, int root, int *tree, int ndim, int size, float *data, int nn, float *points, int *res)
{
	int i = threadIdx.x + blockIdx.x * CUDA_THREAD;

	float p[3] = { points[i * 3 + 0], points[i * 3 + 1], points[i * 3 + 2] };

	// ������ʽ������
	//int *kNeighbors = gpuKN + i * nn;
	//float *kNDistance = gpuKD + i * nn;
	int kNeighbors[10];
	float kNDistance[10];
	int _currentNNode = 0;

	// ������ʽ��·����ջ
	int paths[64];
	int _currentPath = 0;

	// ��¼����·��
	int node = root;
	while (node > -1)
	{
		paths[_currentPath++] = node;

		node = p[tree[node]] <= data[tree[node] * size + node] ? tree[2 * size + node] : tree[3 * size + node];
	}

	kNeighbors[_currentNNode] = -1;
	kNDistance[_currentNNode++] = 9999999;

	// ����·��
	float distance = 0;
	while (_currentPath > 0)
	{
		node = paths[_currentPath-- - 1];

		float sum = 0;
		for (int j = 0; j < ndim; j++)
		{
			sum += (p[j] - data[j * size + node]) * (p[j] - data[j * size + node]);
		}
		distance = sum;

		if (_currentNNode < nn)
		{
			kNeighbors[_currentNNode] = node;
			kNDistance[_currentNNode++] = distance;

			// ���ﵽk���ڵ�󣬽�����������
			if (_currentNNode == nn)
			{
				for (int j = _currentNNode / 2 - 1; j >= 0; j--)
				{
					int parent = j;

					for (int son = j * 2 + 1; son <= _currentNNode; son = son * 2 + 1)
					{
						if (son + 1 < _currentNNode && kNDistance[son] < kNDistance[son + 1])
							son++;

						if (kNDistance[parent] < kNDistance[son])  // ������ڵ�С���ӽڵ㣬�򽻻�
						{
							float tempD = kNDistance[parent];
							int tempI = kNeighbors[parent];
							kNDistance[parent] = kNDistance[son];
							kNeighbors[parent] = kNeighbors[son];
							kNDistance[son] = tempD;
							kNeighbors[son] = tempI;
						}

						parent = son;
					}
				}
			}
		}
		else
		{
			if (distance < kNDistance[0])
			{
				// pop
				kNeighbors[0] = kNeighbors[_currentNNode - 1];
				kNDistance[0] = kNDistance[_currentNNode - 1];

				// ɾ���Ѷ���Ҫ�ع�����
				int parent = 0;
				int son = parent * 2 + 1;
				for (; son < _currentNNode - 1; son = son * 2 + 1)
				{
					if (son + 1 < _currentNNode - 1 && kNDistance[son] < kNDistance[son + 1])
						son++;

					if (kNDistance[parent] < kNDistance[son])  // ������ڵ�С���ӽڵ㣬�򽻻�
					{
						float tempD = kNDistance[parent];
						int tempI = kNeighbors[parent];
						kNDistance[parent] = kNDistance[son];
						kNeighbors[parent] = kNeighbors[son];
						kNDistance[son] = tempD;
						kNeighbors[son] = tempI;
					}

					parent = son;
				}

				// push
				son = _currentNNode - 1;
				parent = (son - 1) / 2;
				while (son != 0 && distance > kNDistance[parent])
				{
					kNeighbors[son] = kNeighbors[parent];
					kNDistance[son] = kNDistance[parent];
					son = parent;
					parent = (son - 1) / 2;
				}

				kNDistance[son] = distance;
				kNeighbors[son] = node;
			}
		}

		if (tree[2 * size + node] + tree[3 * size + node] > -2)
		{
			int dim = tree[node];
			if (p[dim] > data[dim * size + node])
			{
				if (p[dim] - data[dim * size + node] < kNDistance[0] && tree[2 * size + node] > -1)
				{
					int reNode = tree[2 * size + node];
					while (reNode > -1)
					{
						paths[_currentPath++] = reNode;

						reNode = p[tree[reNode]] <= data[tree[reNode] * size + reNode] ? tree[2 * size + reNode] : tree[3 * size + reNode];
					}
				}
			}
			else
			{
				if (data[dim * size + node] - p[dim] < kNDistance[0] && tree[3 * size + node] > -1)
				{
					int reNode = tree[3 * size + node];
					while (reNode > -1)
					{
						paths[_currentPath++] = reNode;

						reNode = p[tree[reNode]] <= data[tree[reNode] * size + reNode] ? tree[2 * size + reNode] : tree[3 * size + reNode];
					}
				}
			}
		}
	}

	// ���ж�����
	for (int j = _currentNNode - 1; j > 0; j--)
	{
		int tempI = kNeighbors[0];
		float tempD = kNDistance[0];
		kNeighbors[0] = kNeighbors[j];
		kNDistance[0] = kNDistance[j];
		kNeighbors[j] = tempI;
		kNDistance[j] = tempD;

		int parent = 0;
		int son = parent * 2 + 1;
		for (; son < j; son = parent * 2 + 1)
		{
			if (son + 1 < j && kNDistance[son] < kNDistance[son + 1])
				son++;

			if (kNDistance[parent] < kNDistance[son])
			{
				tempD = kNDistance[parent];
				tempI = kNeighbors[parent];
				kNDistance[parent] = kNDistance[son];
				kNeighbors[parent] = kNeighbors[son];
				kNDistance[son] = tempD;
				kNeighbors[son] = tempI;
			}

			parent = son;
		}
	}

	int j = nn;
	while (j != 0)
	{
		j--;
		res[i * nn + j] = kNeighbors[j];
	}
}


int gpuSearchKnnKdtree(int root, int ndim, int size, int nn, float *points, int *res)
{
	if (!points || hipMemcpy(_gpuSearchPoint, points, ALLTHREADS * ndim * sizeof(float), hipMemcpyHostToDevice) != hipSuccess)
	{
		return 1;
	}

	gpuSearchKernal<<<CUDA_BLOCK, CUDA_THREAD>>>(_gpuKNeighbors, _gpuKDistances, root, _gpuKdTreeIndex, 
		ndim, size, _gpuKdTreeData, nn, _gpuSearchPoint, _gpuSearchResult);

	if (hipDeviceSynchronize() != hipSuccess)
	{
		return 1;
	}

	if (hipMemcpy(res, _gpuSearchResult, ALLTHREADS * nn * sizeof(int), hipMemcpyDeviceToHost) != hipSuccess)
	{
		return 1;
	}

	return 0;
}

int gpuFreeCuda()
{
	hipFree(_gpuKdTreeIndex);
	hipFree(_gpuKdTreeData);
	hipFree(_gpuSearchPoint);
	hipFree(_gpuSearchResult);

	hipFree(_gpuKNeighbors);
	hipFree(_gpuKDistances);

	return 0;
}

int getCudaDeviceCount()
{
	int count;
	hipGetDeviceCount(&count);

	return count;
}

int getCudaDeviceNames(int i, char name[])
{
	hipDeviceProp_t prop;
	if (hipGetDeviceProperties(&prop, i) == hipSuccess)
	{
		int n = strlen(prop.name);
		for (int i = 0; i < n; i++)
		{
			name[i] = prop.name[i];
		}
		name[n] = '\0';

		return 0;
	}

	return 1;
}

int setCudaStatus(int ndevice)
{
	if (hipSetDevice(ndevice) == hipSuccess)
	{
		return 0;
	}

	return 1;
}